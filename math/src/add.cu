#include "hip/hip_runtime.h"
#include "add.cuh"

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int add(int a, int b)
{
    int result, *localResult;

    hipMalloc((void**)&localResult, sizeof(int));
    add<<<1,1>>>(a, b, localResult);
    hipMemcpy(&result, localResult, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(localResult);

    return result;
}
