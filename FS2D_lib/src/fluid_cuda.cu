#include "hip/hip_runtime.h"
﻿#include "fluid.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cudart_platform.h"
#include ""

#include <algorithm>
#include <cstdlib>
#include <iostream>

#define sci(x) static_cast<int>(x)
#define scu(x) static_cast<uint8_t>(x)

struct Coord2D
{
	float x = 0.0f, y = 0.0f;

	__device__ Coord2D operator-(const Coord2D& _other);
	__device__ Coord2D operator-();
	__device__ Coord2D operator+(const Coord2D& _other);
	__device__ Coord2D operator*(float _number);

	__device__ Coord2D operator-() const;
	__device__ Coord2D operator*(float _number) const;
};

struct Coord2DI
{
	int x = 0, y = 0;
};

struct RGBColor
{
	float R = 0.0f, G = 0.0f, B = 0.0f;

	RGBColor() = default;
	__host__ __device__ RGBColor(float _R, float _G, float _B);

	__host__ __device__ RGBColor operator+(const RGBColor& _other);
	__host__ __device__ RGBColor operator*(float _num);

	__host__ __device__ RGBColor operator*(float _num) const;
};

struct Particle
{
	Coord2D u;
	RGBColor color;
};

static Config globalConfig;

static const int globalColorArraySize = 7;
static RGBColor globalColorArray[globalColorArraySize], globalCurrentColor;
static uint8_t* globalColorField;

static Particle *globalCurrField, *globalPrevField;
static unsigned int globalXSize, globalYSize;

static float *globalPrevPressure, *globalCurrPressure, *globalVorticityField;
static float elapsedTime = 0.0f, timeSincePress = 0.0f;

__device__ Coord2D Coord2D::operator-(const Coord2D& _other)
{
	return { x - _other.x, y - _other.y };
}

__device__ Coord2D Coord2D::operator-()
{
	return { -x, -y };
}

__device__ Coord2D Coord2D::operator+(const Coord2D& _other)
{
	return { x + _other.x, y + _other.y };
}

__device__ Coord2D Coord2D::operator*(float _num)
{
	return { x * _num, y * _num };
}

__device__ Coord2D Coord2D::operator-() const
{
	return { -x, -y };
}

__device__ Coord2D Coord2D::operator*(float _num) const
{
	return { x * _num, y * _num };
}

__host__ __device__ RGBColor::RGBColor(float _R, float _G, float _B)
	: R(min(255.0f, max(0.0f, _R))), G(min(255.0f, max(0.0f, _G))), B(min(255.0f, max(0.0f, _B))) {}

__host__ __device__ RGBColor RGBColor::operator+(const RGBColor& _other)
{
	return { R + _other.R, G + _other.G, B + _other.B };
}

__host__ __device__ RGBColor RGBColor::operator*(float _num)
{
	return { R * _num, G * _num, B * _num };
}

__host__ __device__ RGBColor RGBColor::operator*(float _num) const
{
	return { R * _num, G * _num, B * _num };
}

__device__ Coord2D computeU(Particle* _particleField, size_t _xSize, size_t _ySize, const Coord2DI& _coordV, const Coord2D& _coordB, float _alpha, float _beta)
{
	int posX = _coordV.x, posY = _coordV.y;
	auto f = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _particleField[_y * _xSize + _x].u : -_coordB; };
	return (f(posX, posY - 1) + f(posX, posY + 1) + f(posX + 1, posY) + f(posX - 1, posY) + _coordB * _alpha) * (1.0f / _beta);
}

__device__ float computeP(float* _pressureField, size_t _xSize, size_t _ySize, const Coord2DI& _coord, float _B, float _alpha, float _beta)
{
	int posX = _coord.x, posY = _coord.y;
	auto f = [&](int x, int y) { return (x < _xSize && x >= 0 && y < _ySize && y >= 0) ? _pressureField[y * _xSize + x] : -_pressureField[posY * _xSize + posX]; };
	return (f(posX, posY - 1) + f(posX, posY + 1) + f(posX - 1, posY) + f(posX + 1, posY) + _alpha * _B) * (1.0f / _beta);
}

__device__ RGBColor computeC(Particle* _colorField, size_t _xSize, size_t _ySize, const Coord2DI& _coordPos, const RGBColor& _colorB, float _alpha, float _beta)
{
	int x = _coordPos.x, y = _coordPos.y;
	auto f = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _colorField[_y * _xSize + _x].color : RGBColor(); };
	return (f(x, y - 1) + f(x, y + 1) + f(x - 1, y) + f(x + 1, y) + _colorB * _alpha) * (1.0f / _beta);
}

__device__ Particle interpolate(const Coord2D& _pos, Particle* _particleField, size_t _xSize, size_t _ySize)
{
	int x1 = sci(_pos.x), y1 = sci(_pos.y),
		x2 = sci(_pos.x) + 1, y2 = sci(_pos.y) + 1;

	auto clamp = [](int _val, int _maxVal) { return min(_maxVal - 1, max(0, _val)); };

	Particle q1 = _particleField[clamp(y1, _ySize) * _xSize + clamp(x1, _xSize)],
		 q2 = _particleField[clamp(y2, _ySize) * _xSize + clamp(x1, _xSize)],
		 q3 = _particleField[clamp(y1, _ySize) * _xSize + clamp(x2, _xSize)],
		 q4 = _particleField[clamp(y2, _ySize) * _xSize + clamp(x2, _xSize)];

	float t1 = (x2 - _pos.x) / (x2 - x1), t2 = (_pos.x - x1) / (x2 - x1),
		  t3 = (y2 - _pos.y) / (y2 - y1), t4 = (_pos.y - y1) / (y2 - y1);

	return { (q1.u * t1 + q3.u * t2) * t3 + (q2.u * t1 + q4.u * t2) * t4,
		 (q2.color * t1 + q4.color * t2) * t3 + (q2.color * t1 + q4.color * t2) * t4 };
}

__device__ float divergency(Particle* _particleField, size_t _xSize, size_t _ySize, const Coord2DI& _coordPos)
{
	int posX = _coordPos.x, posY = _coordPos.y;
	Particle& C = _particleField[posY * _xSize + posX];
	auto f = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _particleField[_y * _xSize + _x].u.x : -C.u.x; };
	auto g = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _particleField[_y * _xSize + _x].u.y : -C.u.y; };
	return (f(posX + 1, posY) - f(posX - 1, posY) + g(posX, posY + 1) - g(posX, posY - 1)) / 2.0f;
}

__device__ Coord2D gradient(float* _field, size_t _xSize, size_t _ySize, const Coord2DI& _coordPos)
{
	int posX = _coordPos.x, posY = _coordPos.y;
	auto f = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _field[_y * _xSize + _x] : _field[posY * _xSize + posX]; };
	return { (f(posX + 1, posY) - f(posX - 1, posY)) * 0.5f, (f(posX, posY + 1) - f(posX, posY - 1)) / 2.0f };
}

__device__ Coord2D absGradient(float* _field, size_t _xSize, size_t _ySize, const Coord2DI& _coordPos)
{
	int posX = _coordPos.x, posY = _coordPos.y;
	auto f = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _field[_y * _xSize + _x] : _field[posY * _xSize + posX]; };
	return { (abs(f(posX + 1, posY)) - abs(f(posX - 1, posY))) / 2.0f, (abs(f(posX, posY + 1)) - abs(f(posX, posY - 1))) / 2.0f };
}

__device__ float curl(Particle* _particleField, size_t _xSize, size_t _ySize, const Coord2DI& _coordPos)
{
	int posX = _coordPos.x, posY = _coordPos.y;
	auto f = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _particleField[_y * _xSize + _x].u.x : -_particleField[posY * _xSize + posX].u.x; };
	auto g = [&](int _x, int _y) { return (_x < _xSize && _x >= 0 && _y < _ySize && _y >= 0) ? _particleField[_y * _xSize + _x].u.y : -_particleField[posY * _xSize + posX].u.y; };
	return ((g(posX, posY + 1) - g(posX, posY - 1)) - (f(posX + 1, posY) - f(posX - 1, posY))) / 2.0f;
}

__global__ void advect(Particle* _particleCurrField, Particle* _particlePrevField, size_t _xSize, size_t _ySize, float _densityDiffusion, float _dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	float decay = 1.0f / (1.0f + _densityDiffusion * _dt);

	Coord2D pos = { static_cast<float>(x), static_cast<float>(y) };
	Particle &prevP = _particlePrevField[y * _xSize + x],
			 p = interpolate(pos - prevP.u * _dt, _particlePrevField, _xSize, _ySize);

	_particleCurrField[y * _xSize + x] = { p.u * decay, { pow(p.color.R, 1.005f) * decay, pow(p.color.G, 1.005f) * decay, pow(p.color.B, 1.005f) * decay } };
}

__global__ void paint(Particle* _particleField, uint8_t* _colorField, size_t _xSize, size_t _ySize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	float R = _particleField[y * _xSize + x].color.R, G = _particleField[y * _xSize + x].color.G, B = _particleField[y * _xSize + x].color.B;

	_colorField[4 * (y * _xSize + x) + 0] = scu(min(255.0f, 255.0f * R));
	_colorField[4 * (y * _xSize + x) + 1] = scu(min(255.0f, 255.0f * G));
	_colorField[4 * (y * _xSize + x) + 2] = scu(min(255.0f, 255.0f * B));
	_colorField[4 * (y * _xSize + x) + 3] = scu(255.0f);
}

__global__ void project(Particle* _particleField, size_t _xSize, size_t _ySize, float* _fieldP)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	Coord2D& u = _particleField[y * _xSize + x].u;
	u = u - gradient(_fieldP, _xSize, _ySize, { x, y });
}

__global__ void computeVelocity(Particle* _particleCurrField, Particle* _particlePrevField, size_t _xSize, size_t _ySize, float _velocityDiffusion, float _dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	float alpha = _velocityDiffusion * _velocityDiffusion / _dt;
	_particleCurrField[y * _xSize + x].u = computeU(_particlePrevField, _xSize, _ySize, { x, y }, _particlePrevField[y * _xSize + x].u, alpha, 4.0f + alpha);
}

__global__ void computePressure(Particle* _particleCurrField, size_t _xSize, size_t _ySize, float* _currP, float* _prevP, float _pressure)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	_currP[y * _xSize + x] = computeP(_prevP, _xSize, _ySize, { x, y }, divergency(_particleCurrField, _xSize, _ySize, { x, y }), -_pressure * _pressure, 4.0f);
}

__global__ void computeColor(Particle* _particleCurrField, Particle* _particlePrevField, size_t _xSize, size_t _ySize, float _colorDiffusion, float _dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	float alpha = _colorDiffusion * _colorDiffusion / _dt;
	_particleCurrField[y * _xSize + x].color = computeC(_particlePrevField, _xSize, _ySize, { x, y }, _particlePrevField[y * _xSize + x].color, alpha, 4.0f + alpha);
}

__global__ void computeVorticity(Particle* _particleField, float* _fieldV, size_t _xSize, size_t _ySize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	_fieldV[y * _xSize + x] = curl(_particleField, _xSize, _ySize, { x, y });
}

__global__ void applyForce(Particle* _particleField, size_t _xSize, RGBColor _color, Coord2D _coordF, Coord2DI _pos, int _r, float _dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	float e = expf(-((x - _pos.x) * (x - _pos.x) + (y - _pos.y) * (y - _pos.y)) / _r);
	Particle& p = _particleField[y * _xSize + x];

	_color = _color * e + p.color;
	p = { p.u + _coordF * _dt * e, { _color.R, _color.G, _color.B } };
}

__global__ void applyVorticity(Particle* _particleCurrField, Particle* _particlePrevField, float* _fieldV, size_t _xSize, size_t _ySize, float _vorticity, float _dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;

	Particle &pOld = _particlePrevField[y * _xSize + x], &pNew = _particleCurrField[y * _xSize + x];

	Coord2D v = absGradient(_fieldV, _xSize, _ySize, { x, y });
	v.y *= -1.0f;
	Coord2D vNorm = v * (1.0f / (sqrtf(v.x * v.x + v.y * v.y) + 1e-5f));

	pNew = pOld;
	pNew.u = pNew.u + (vNorm * _fieldV[y * _xSize + x] * _vorticity) * _dt;
}

__global__ void applyBloom(uint8_t* _colorField, size_t _xSize, Coord2DI _pos, int _radius, float _bloomIntense)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y, pos = 4 * (y * _xSize + x);
	float e = _bloomIntense * expf(-((x - _pos.x) * (x - _pos.x) + (y - _pos.y) * (y - _pos.y) + 1.0f) / (_radius * _radius));
	uint8_t R = _colorField[pos + 0], G = _colorField[pos + 1], B = _colorField[pos + 2], maxval = max(R, max(G, B));

	_colorField[pos + 0] = min(255.0f, R + maxval * e);
	_colorField[pos + 1] = min(255.0f, G + maxval * e);
	_colorField[pos + 2] = min(255.0f, B + maxval * e);
}

// Export functions
void setConfig(float _velocityDiffusion, float _colorDiffusion, float _densityDiffusion, float _pressure, float _vorticity,
			   float _forceScale, float _bloomIntense, float _dt, int _radius, int _velocityIterations, int _pressureIterations,
			   int _xThreads, int _yThreads, bool _bloomEnabled, bool _colorful, bool _parallel, unsigned char _r, unsigned char _g, unsigned char _b)
{
	globalConfig.velocityDiffusion = _velocityDiffusion;
	globalConfig.colorDiffusion = _colorDiffusion;
	globalConfig.densityDiffusion = _densityDiffusion;
	globalConfig.pressure = _pressure;
	globalConfig.vorticity = _vorticity;
	globalConfig.forceScale = _forceScale;
	globalConfig.bloomIntense = _bloomIntense;
	globalConfig.dt = _dt;
	globalConfig.radius = _radius;
	globalConfig.velocityIterations = _velocityIterations;
	globalConfig.pressureIterations = _pressureIterations;
	globalConfig.xThreads = _xThreads;
	globalConfig.yThreads = _yThreads;
	globalConfig.bloomEnabled = _bloomEnabled;
	globalConfig.bColorful = _colorful;
	globalConfig.bParallel = _parallel;
	globalConfig.color[0] = _r;
	globalConfig.color[1] = _g;
	globalConfig.color[2] = _b;

	globalColorArray[0] = globalConfig.bColorful ? RGBColor({ 1.0f, 0.0f, 0.0f }) : RGBColor({ globalConfig.color[0] / 255.f, globalConfig.color[1] / 255.f, globalConfig.color[2] / 255.f });
}

void initialization(int _x, int _y)
{
	setConfig();

	globalColorArray[0] = globalConfig.bColorful ? RGBColor({ 1.0f, 0.0f, 0.0f }) : RGBColor({ globalConfig.color[0] / 255.f, globalConfig.color[1] / 255.f, globalConfig.color[2] / 255.f });
	globalColorArray[1] = { 0.0f, 1.0f, 0.0f };
	globalColorArray[2] = { 1.0f, 0.0f, 1.0f };
	globalColorArray[3] = { 1.0f, 1.0f, 0.0f };
	globalColorArray[4] = { 0.0f, 1.0f, 1.0f };
	globalColorArray[5] = { 1.0f, 0.0f, 1.0f };
	globalColorArray[6] = { 1.0f, 0.5f, 0.3f };

	globalCurrentColor = globalConfig.bColorful ? globalColorArray[rand() % globalColorArraySize] : globalColorArray[0];

	globalXSize = _x, globalYSize = _y;

	hipSetDevice(0);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		std::cout << hipGetErrorName(error) << std::endl;
		std::abort();
	}

	size_t size = globalXSize * globalYSize;

	hipMalloc(&globalColorField, 4 * size * sizeof(uint8_t));
	hipMalloc(&globalPrevField, size * sizeof(Particle));
	hipMalloc(&globalCurrField, size * sizeof(Particle));
	hipMalloc(&globalPrevPressure, size * sizeof(float));
	hipMalloc(&globalCurrPressure, size * sizeof(float));
	hipMalloc(&globalVorticityField, size * sizeof(float));
}

void finalization()
{
	hipFree(globalColorField);
	hipFree(globalPrevField);
	hipFree(globalCurrField);
	hipFree(globalPrevPressure);
	hipFree(globalCurrPressure);
	hipFree(globalVorticityField);
}

void compute(uint8_t* _resultField, int _prevX, int _prevY, int _currX, int _currY, bool _isPressed)
{
	dim3 threadsPerBlock(globalConfig.bParallel ? globalConfig.xThreads : 1, globalConfig.bParallel ? globalConfig.yThreads : 1);
	dim3 numBlocks(globalXSize / threadsPerBlock.x, globalYSize / threadsPerBlock.y);


	computeVorticity<<<numBlocks, threadsPerBlock>>>(globalPrevField, globalVorticityField, globalXSize, globalYSize);
	applyVorticity<<<numBlocks, threadsPerBlock>>>(globalCurrField, globalPrevField, globalVorticityField, globalXSize, globalYSize, globalConfig.vorticity, globalConfig.dt);
	std::swap(globalPrevField, globalCurrField);

	for (int i = 0; i < globalConfig.velocityIterations; ++i)
	{
		computeVelocity<<<numBlocks, threadsPerBlock>>>(globalCurrField, globalPrevField, globalXSize, globalYSize, globalConfig.velocityDiffusion, globalConfig.dt);
		computeColor<<<numBlocks, threadsPerBlock>>>(globalCurrField, globalPrevField, globalXSize, globalYSize, globalConfig.colorDiffusion, globalConfig.dt);
		std::swap(globalCurrField, globalPrevField);
	}

	if (_isPressed)
	{
		timeSincePress = 0.0f;
		elapsedTime += globalConfig.dt;

		float w = elapsedTime - sci(elapsedTime);
		globalCurrentColor = globalConfig.bColorful ? globalColorArray[sci(elapsedTime) % globalColorArraySize] * (1 - w) + globalColorArray[sci((elapsedTime) + 1) % globalColorArraySize] * w : globalColorArray[0];
		applyForce<<<numBlocks, threadsPerBlock>>>(globalPrevField, globalXSize, globalCurrentColor,
							   { (_currX - _prevX) * globalConfig.forceScale, (_currY - _prevY) * globalConfig.forceScale }, { _currX, _currY }, globalConfig.radius, globalConfig.dt);
	}
	else
	{
		timeSincePress += globalConfig.dt;
	}

	for (int i = 0; i < globalConfig.pressureIterations; ++i)
	{
		computePressure<<<numBlocks, threadsPerBlock>>>(globalPrevField, globalXSize, globalYSize, globalCurrPressure, globalPrevPressure, globalConfig.pressure);
		std::swap(globalPrevPressure, globalCurrPressure);
	}

	project<<<numBlocks, threadsPerBlock>>>(globalPrevField, globalXSize, globalYSize, globalPrevPressure);
	hipMemset(globalPrevPressure, 0, globalXSize * globalYSize * sizeof(float));

	advect<<<numBlocks, threadsPerBlock>>>(globalCurrField, globalPrevField, globalXSize, globalYSize, globalConfig.densityDiffusion, globalConfig.dt);
	std::swap(globalCurrField, globalPrevField);

	paint<<<numBlocks, threadsPerBlock>>>(globalPrevField, globalColorField, globalXSize, globalYSize);

	if (globalConfig.bloomEnabled && timeSincePress < 5.0f)
	{
		applyBloom<<<numBlocks, threadsPerBlock>>>(globalColorField, globalXSize, { _currX, _currY }, globalConfig.radius, globalConfig.bloomIntense);
	}

	hipMemcpy(_resultField, globalColorField, globalXSize * globalYSize * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		std::cout << hipGetErrorName(error) << std::endl;
	}
}